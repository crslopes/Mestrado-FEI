#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <limits>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <omp.h> // adiciona biblioteca do Open MP

using namespace std;
//-------------------------------------------------------------------
//      PEL_216 2�. Semestre de 2019
//      Prof: Dr Reinaldo Bianchi
//      Aluno: Cristiano Lopes Moreira
//      RA: 119103-0
//
//      Integral Numerica CUDA
//
//-------------------------------------------------------------------

using std::cout;
using std::endl;


const long mTHREADS = 32; // tamanho da thread


//  kernel cuda
__global__ void integralMontecarlo(long *totals, double*THREADS, long int*LOOPS) {
	// Define some shared memory: all threads in this block
	__shared__ long counter[mTHREADS];

	//ID da thread
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// Initialize RNG
	hiprandState_t rng;
	hiprand_init(clock64(), tid, 0, &rng);

	// Initialize the counter
	counter[threadIdx.x] = 0;

	//loop
//	for (long int  i = 0; i < *LOOPS; i++) {
//
//		float x = hiprand_uniform(&rng); // Random x position in [0,1]
//		float y = hiprand_uniform(&rng); // Random y position in [0,1]
//		counter[threadIdx.x] += 1 - int(x * x + y * y); // Hit test

//	}


	for (long i = 0; i < *LOOPS; i++) {
		float x = 1.0 + 3.0 * hiprand_uniform(&rng); // Random x entre [1,4]
		float y = -3.0 + 7.0 * hiprand_uniform(&rng); // Random y entre [-3,4]
		float z = -1.0 + 2.0 * hiprand_uniform(&rng); // Random y entre [-1,1]
		//equa�ao do toroide para contabilizar ocorrencias sorteadas no toroide
		if ((pow(z, 2) + pow(sqrt(pow(x, 2) + pow(y, 2)) - 3, 2)) <= 1 && (x >= 1) && (y >= -3)) {
			counter[threadIdx.x] += 1;
		}
	}


	// a thread 0 realiza o agrupamento dos resultados do bloco de processo
	if (threadIdx.x == 0) {
		// zera o contador dos resultados
		totals[blockIdx.x] = 0;
		// acumula os resultados
		for (int  i = 0; i < mTHREADS; i++) {
			totals[blockIdx.x] += counter[i];
		}
	}
}



int main(int argc, char** argv) {
	int numDev;
	long NBLOCKS = 5120;
	double threads = mTHREADS;
	long int loops = 5000;
	double* dTHREADS;
	long int* dLOOPS;

	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		cout << "GPU nao encontrada\n";
		return 1;
	}

	if (argc < 2) {
		printf("Entre com o numero de interacoes e numero de cores  IntegralNumericaGPU <interacoes> <cores>\n");
		exit(1);
	}

	NBLOCKS = (double)atoi(argv[2]);

	long int tests = strtol(argv[1], NULL, 10);  //(long int)(argv[1]);

	loops = tests / NBLOCKS / mTHREADS/numDev;

	cout << "Inicializa com " << NBLOCKS << " Cuda Blocks, " << threads << " threads,  "<< tests << " interacoes e "<< numDev << " GPUs" << endl;


	long* hOut[2], * dOut[2];
	hOut[0] = new long[NBLOCKS]; // memoria do servidor
	hOut[1] = new long[NBLOCKS]; // memoria do servidor


	 
	hipSetDevice(0);

	// aloca�ao de memoria

	hipMalloc(&dOut[0], sizeof(long) * NBLOCKS); // memoria da GPU

	hipMalloc(&dTHREADS, sizeof(double)); // memoria da GPU
	hipMemcpy(dTHREADS, &threads, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc(&dLOOPS, sizeof(long int)); // memoria da GPU
	hipMemcpy(dLOOPS, &loops, sizeof(long int), hipMemcpyHostToDevice);

	// Launch kernel
	integralMontecarlo << <NBLOCKS, threads >> > (dOut[0], dTHREADS, dLOOPS);

	hipSetDevice(1);
	hipMalloc(&dOut[1], sizeof(long) * NBLOCKS); // memoria da GPU

	hipMalloc(&dTHREADS, sizeof(double) ); // memoria da GPU
	hipMemcpy(dTHREADS, &threads, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc(&dLOOPS, sizeof(long int)); // memoria da GPU
	hipMemcpy(dLOOPS, &loops, sizeof(long int), hipMemcpyHostToDevice);

	integralMontecarlo << <NBLOCKS, threads >> > (dOut[1], dTHREADS, dLOOPS);

	// Copy back memory used on device and free
	hipMemcpy(hOut[0], dOut[0], sizeof(long) * NBLOCKS, hipMemcpyDeviceToHost);
	hipMemcpy(hOut[1], dOut[1], sizeof(long) * NBLOCKS, hipMemcpyDeviceToHost);
	hipFree(dOut[0]);
	hipFree(dOut[1]);

	// Compute total hits
	long total = 0;

#pragma omp parallel for private(i) reduction (+:total)

	for (int i = 0; i < NBLOCKS; i++) {
		total += hOut[0][i] + hOut[1][i];
	}


	std::cout.precision(150);


//	long tests = NBLOCKS * LOOPS * THREADS * 2;
	cout << "Volume aproximado do Toroide, Metodo de Monte Carlo, com  " << tests << " testes aleatorios\n";

	cout << "Volume Toroide ~= " << (4.0 - 1.0) * (4.0 - (-3.0)) * (1.0 - (-1.0)) * (double)total / (double)tests << endl;

	return 0;


//	cout << "PI ~= " << 4.0 * total /tests << endl;

//	return 0;
}
